#include "hip/hip_runtime.h"
/*! \file CudaShadingContext_cuda.cu
 *  \author Jared Hoberock
 *  \brief CUDA implementation of CudaShadingContext class.
 */

// XXX remove this when exceptions are enabled in nvcc
#define BOOST_NO_EXCEPTIONS

#include <stdcuda/stride_cast.h>
#include "CudaShadingContext.h"

#include "CudaScatteringDistributionFunction.h"

using namespace stdcuda;

struct Parameters
{
  const CudaScatteringDistributionFunction *f;
  const CudaDifferentialGeometry *dg;
  int dgStride;
  const float3 *u;
  int uStride;
  float3 *s;
  int sStride;
  float3 *wo;
  int woStride;
  float *pdf;
  int pdfStride;
  bool *delta;
  int deltaStride;
}; // end Parameters

void __global__ sampleUnidirectionalKernel(const Parameters p)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // unpack the streams (don't cross)
  // XXX this shit doesn't compile, thanks cuda
  //const CudaDifferentialGeometry *diffG = stride_cast(p.dg, i, p.dgStride);
  //float3 x = *stdcuda::stride_cast<float3>(u, i, uStride);
  //float3 &result = *stride_cast(wo, i, woStride);
  //float &pdfResult = *stride_cast(pdf, i, pdfStride);
  //bool &deltaResult = *stride_cast(delta, i, deltaStride);

  const char *ptr = reinterpret_cast<const char*>(p.dg) + i*p.dgStride;
  const CudaDifferentialGeometry &dg = *reinterpret_cast<const CudaDifferentialGeometry*>(ptr);

  ptr = reinterpret_cast<const char*>(p.u) + i*p.uStride;
  float3 u = *reinterpret_cast<const float3*>(ptr);

  char *temp = reinterpret_cast<char*>(p.s) + i*p.sStride;
  float3 &s = *reinterpret_cast<float3*>(temp);

  temp = reinterpret_cast<char*>(p.wo) + i*p.woStride;
  float3 &wo = *reinterpret_cast<float3*>(temp);

  temp = reinterpret_cast<char*>(p.pdf) + i*p.pdfStride;
  float &pdf = *reinterpret_cast<float*>(temp);

  temp = reinterpret_cast<char*>(p.delta) + i*p.deltaStride;
  bool &delta = *reinterpret_cast<bool*>(temp);

  const CudaScatteringDistributionFunction &f = p.f[i];

  // sample
  f.sample(dg, u.x, u.y, u.z, s, wo, pdf, delta);
} // end sampleUnidirectionalKernel()

void CudaShadingContext
  ::sampleUnidirectionalScattering(const device_ptr<const CudaScatteringDistributionFunction> &f,
                                   const device_ptr<const CudaDifferentialGeometry> &dg,
                                   const size_t dgStride,
                                   const device_ptr<const float3> &u,
                                   const size_t uStride,
                                   const device_ptr<float3> &s,
                                   const size_t sStride,
                                   const device_ptr<float3> &wo,
                                   const size_t woStride,
                                   const device_ptr<float> &pdf,
                                   const size_t pdfStride,
                                   const device_ptr<bool> &delta,
                                   const size_t deltaStride,
                                   const size_t n)
{
  unsigned int BLOCK_SIZE = 192;
  unsigned int gridSize = n / BLOCK_SIZE;

  Parameters p = {f, dg, dgStride, u, uStride, s, sStride, wo, woStride, pdf, pdfStride, delta, deltaStride};

  if(gridSize)
    sampleUnidirectionalKernel<<<gridSize,BLOCK_SIZE>>>(p);
  if(n%BLOCK_SIZE)
  {
    Parameters p = {f + gridSize*BLOCK_SIZE,
                    stride_cast(dg.get(), gridSize*BLOCK_SIZE, dgStride),
                    dgStride,
                    stride_cast(u.get(),  gridSize*BLOCK_SIZE, uStride),
                    uStride,
                    stride_cast(s.get(),  gridSize*BLOCK_SIZE, sStride),
                    sStride,
                    stride_cast(wo.get(), gridSize*BLOCK_SIZE, woStride),
                    woStride,
                    stride_cast(pdf.get(), gridSize*BLOCK_SIZE, pdfStride),
                    pdfStride,
                    stride_cast(delta.get(), gridSize*BLOCK_SIZE, deltaStride),
                    deltaStride};
    sampleUnidirectionalKernel<<<1,n%BLOCK_SIZE>>>(p);
  } // end if
} // end CudaDifferentialGeometry::sampleUnidirectionalScattering()

struct BiParameters
{
  const CudaScatteringDistributionFunction *f;
  const float3 *wo;
  int woStride;
  const CudaDifferentialGeometry *dg;
  int dgStride;
  const float3 *u;
  int uStride;
  const bool *stencil;
  float3 *s;
  int sStride;
  float3 *wi;
  int wiStride;
  float *pdf;
  int pdfStride;
  bool *delta;
  int deltaStride;
  unsigned int *component;
  int componentStride;
}; // end Parameters

void __global__ sampleBidirectionalKernel(const BiParameters p)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // sample
  if(p.stencil[i])
  {
    const char *ptr = reinterpret_cast<const char*>(p.dg) + i*p.dgStride;
    const CudaDifferentialGeometry &dg = *reinterpret_cast<const CudaDifferentialGeometry*>(ptr);

    ptr = reinterpret_cast<const char*>(p.wo) + i*p.woStride;
    const float3 &wo = *reinterpret_cast<const float3*>(ptr);

    ptr = reinterpret_cast<const char*>(p.u) + i*p.uStride;
    float3 u = *reinterpret_cast<const float3*>(ptr);

    char *temp = reinterpret_cast<char*>(p.s) + i*p.sStride;
    float3 &s = *reinterpret_cast<float3*>(temp);

    temp = reinterpret_cast<char*>(p.wi) + i*p.wiStride;
    float3 &wi = *reinterpret_cast<float3*>(temp);

    temp = reinterpret_cast<char*>(p.pdf) + i*p.pdfStride;
    float &pdf = *reinterpret_cast<float*>(temp);

    temp = reinterpret_cast<char*>(p.delta) + i*p.deltaStride;
    bool &delta = *reinterpret_cast<bool*>(temp);

    temp = reinterpret_cast<char*>(p.component) + i*p.componentStride;
    unsigned int &component = *reinterpret_cast<unsigned int*>(temp);

    const CudaScatteringDistributionFunction &f = p.f[i];

    f.sample(wo, dg, u.x, u.y, u.z, s, wi, pdf, delta, component);
  } // end if
} // end sampleUnidirectionalKernel()

void CudaShadingContext
  ::sampleBidirectionalScattering(const device_ptr<const CudaScatteringDistributionFunction> &f,
                                  const device_ptr<const float3> &wo,
                                  const size_t woStride,
                                  const device_ptr<const CudaDifferentialGeometry> &dg,
                                  const size_t dgStride,
                                  const device_ptr<const float3> &u,
                                  const size_t uStride,
                                  const device_ptr<bool> &stencil,
                                  const device_ptr<float3> &s,
                                  const size_t sStride,
                                  const device_ptr<float3> &wi,
                                  const size_t wiStride,
                                  const device_ptr<float> &pdf,
                                  const size_t pdfStride,
                                  const device_ptr<bool> &delta,
                                  const size_t deltaStride,
                                  const device_ptr<unsigned int> &component,
                                  const size_t componentStride,
                                  const size_t n)
{
  unsigned int BLOCK_SIZE = 192;
  unsigned int gridSize = n / BLOCK_SIZE;

  BiParameters p = {f,
                    wo,
                    woStride,
                    dg,
                    dgStride,
                    u,
                    uStride,
                    stencil,
                    s,
                    sStride,
                    wi,
                    wiStride,
                    pdf,
                    pdfStride,
                    delta,
                    deltaStride,
                    component,
                    componentStride};

  if(gridSize)
    sampleBidirectionalKernel<<<gridSize,BLOCK_SIZE>>>(p);
  if(n%BLOCK_SIZE)
  {
    BiParameters p = {f + gridSize*BLOCK_SIZE,
                      stride_cast(wo.get(), gridSize*BLOCK_SIZE, woStride),
                      woStride,
                      stride_cast(dg.get(), gridSize*BLOCK_SIZE, dgStride),
                      dgStride,
                      stride_cast(u.get(),  gridSize*BLOCK_SIZE, uStride),
                      uStride,
                      stencil + gridSize*BLOCK_SIZE,
                      stride_cast(s.get(),  gridSize*BLOCK_SIZE, sStride),
                      sStride,
                      stride_cast(wi.get(), gridSize*BLOCK_SIZE, wiStride),
                      wiStride,
                      stride_cast(pdf.get(), gridSize*BLOCK_SIZE, pdfStride),
                      pdfStride,
                      stride_cast(delta.get(), gridSize*BLOCK_SIZE, deltaStride),
                      deltaStride,
                      stride_cast(component.get(), gridSize*BLOCK_SIZE, componentStride),
                      componentStride};
    sampleBidirectionalKernel<<<1,n%BLOCK_SIZE>>>(p);
  } // end if
} // end CudaDifferentialGeometry::sampleBidirectionalScattering()

