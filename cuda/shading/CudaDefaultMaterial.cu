#include "hip/hip_runtime.h"
/*! \file CudaDefaultMaterial.cu
 *  \author Jared Hoberock
 *  \brief Implementation of CudaDefaultMaterial class.
 */

#define inline inline __host__ __device__

#include "CudaDefaultMaterial.h"
#include "CudaScatteringDistributionFunction.h"
#include "../include/CudaShadingInterface.h"
#include <stdcuda/fill_if.h>
using namespace stdcuda;

#undef inline

const char *CudaDefaultMaterial
  ::getName(void) const
{
  return "CudaDefaultMaterial";
} // end CudaDefaultMaterial::getName()

void CudaDefaultMaterial
  ::evaluateScattering(CudaShadingInterface &context,
                       const device_ptr<const CudaDifferentialGeometry> &dg,
                       const size_t dgStride,
                       const device_ptr<const int> &stencil,
                       const device_ptr<CudaScatteringDistributionFunction> &f,
                       const size_t n) const
{
  CudaScatteringDistributionFunction value;

  // create something white
  context.diffuse(make_float3(1,1,1), value);

  fill_if(f.get(), f.get() + n, stencil.get(), value);
} // end CudaMaterial::evaluateScattering()

