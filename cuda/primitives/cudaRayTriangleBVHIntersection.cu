#include "hip/hip_runtime.h"
/*! \file cudaRayTriangleBVHIntersection.cu
 *  \author Jared Hoberock
 *  \brief Implementation of cudaRayTriangleBVHIntersection function.
 */

#include <stdio.h>

#include "cudaRayTriangleBVHIntersection.h"
#include <waldbikkerintersection/cudaWaldBikkerIntersection.h>
#include <stdcuda/vector_math.h>

inline __device__ bool intersectBox(const float3 &o,
                                    const float3 &invDir,
                                    const float3 &minBounds,
                                    const float3 &maxBounds,
                                    const float &tMin,
                                    const float &tMax)
{
  float3 tMin3, tMax3;
  tMin3 = (minBounds - o) * invDir;
  tMax3 = (maxBounds - o) * invDir;

  float3 tNear3 = fminf(tMin3, tMax3);
  float3 tFar3  = fmaxf(tMin3, tMax3);

  float tNear = fmaxf(fmaxf(tNear3.x, tNear3.y), tNear3.z);
  float tFar  = fminf(fminf( tFar3.x,  tFar3.y),  tFar3.z);

  bool hit = tNear <= tFar;
  return hit && tMax >= tNear && tMin <= tFar;
} // end intersectBox()

__global__ void rbk(const unsigned int NULL_NODE,
                    const unsigned int rootIndex,
                    const float3 *origins,
                    const float3 *directions,
                    const float2 interval,
                    const float4 *minBoundHitIndex,
                    const float4 *maxBoundMissIndex,
                    const float4 *firstVertexDominantAxis,
                    bool *stencil,
                    float4 *timeBarycentricsAndTriangleIndex)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(stencil[i])
  {
    float3 origin = origins[i];
    float3 dir = directions[i];
    float2 myInterval = interval;

    float3 invDir = make_float3(1.0f / dir.x,
                                1.0f / dir.y,
                                1.0f / dir.z);
    unsigned int currentNode = rootIndex;
    bool hit = false;
    bool result = false;
    float t = myInterval.y;;
    unsigned int tri = 0;
    float b1 = -1, b2 = -1;
    float4 minBoundsHit, maxBoundsMiss;
    float4 v0Axis;

    // XXX PERF: it might be possible to eliminate these temporaries
    float tempT, tempB1, tempB2;
    while(currentNode != NULL_NODE)
    {
      minBoundsHit = minBoundHitIndex[currentNode];
      maxBoundsMiss = maxBoundMissIndex[currentNode];

      // leaves (primitives) are listed before interior nodes
      // so bounding boxes occur after the root index
      if(currentNode >= rootIndex)
      {
        hit = intersectBox(origin,
                           invDir,
                           make_float3(minBoundsHit.x,
                                       minBoundsHit.y,
                                       minBoundsHit.z),
                           make_float3(maxBoundsMiss.x,
                                       maxBoundsMiss.y,
                                       maxBoundsMiss.z),
                           interval.x,
                           interval.y);
      } // end if
      else
      {
        v0Axis = firstVertexDominantAxis[currentNode];

        hit = cudaWaldBikkerIntersection
          (origin,
           dir,
           myInterval.x, myInterval.y,
           make_float3(v0Axis.x,
                       v0Axis.y,
                       v0Axis.z),
           make_float3(minBoundsHit.x,
                       minBoundsHit.y,
                       minBoundsHit.z),
           __float_as_int(v0Axis.w),
           maxBoundsMiss.x, maxBoundsMiss.y,
           maxBoundsMiss.z, maxBoundsMiss.w,
           tempT, tempB1, tempB2);
        result |= hit;

        // XXX we could potentially merge t and tMax into a single word
        //     as they serve essentially the same purpose
        if(hit)
        {
          t = tempT;
          myInterval.y = tempT;
          tri = currentNode;
          b1 = tempB1;
          b2 = tempB2;
        } // end if

        // ensure that the miss and hit indices are the same
        // at this point
        maxBoundsMiss.w = minBoundsHit.w;
        hit = false;
      } // end else

      currentNode = hit ? __float_as_int(minBoundsHit.w) : __float_as_int(maxBoundsMiss.w);
    } // end while

    // write results
    stencil[i] = result;
    timeBarycentricsAndTriangleIndex[i] = make_float4(t, b1, b2, __int_as_float(tri));
  } // end if
} // end rbk()

void cudaRayTriangleBVHIntersection(const unsigned int NULL_NODE,
                                    const unsigned int rootIndex,
                                    const float3 *origins,
                                    const float3 *directions,
                                    const float2 &interval,
                                    const float4 *minBoundHitIndex,
                                    const float4 *maxBoundMissIndex,
                                    const float4 *firstVertexDominantAxis,
                                    bool *stencil,
                                    float4 *timeBarycentricsAndTriangleIndex,
                                    const size_t n)
{
  unsigned int BLOCK_SIZE = 192;
  unsigned int gridSize = n/BLOCK_SIZE;

  if(gridSize)
    rbk<<<gridSize,BLOCK_SIZE>>>(NULL_NODE,
                                 rootIndex,
                                 origins,
                                 directions,
                                 interval,
                                 minBoundHitIndex,
                                 maxBoundMissIndex,
                                 firstVertexDominantAxis,
                                 stencil,
                                 timeBarycentricsAndTriangleIndex);
  if(n%BLOCK_SIZE)
    rbk<<<1,n%BLOCK_SIZE>>>(NULL_NODE,
                            rootIndex,
                            origins + gridSize*BLOCK_SIZE,
                            directions + gridSize*BLOCK_SIZE,
                            interval,
                            minBoundHitIndex,
                            maxBoundMissIndex,
                            firstVertexDominantAxis,
                            stencil + gridSize*BLOCK_SIZE,
                            timeBarycentricsAndTriangleIndex + gridSize*BLOCK_SIZE);
} // end cudaRayTriangleBVHIntersection()

__global__ void rbk(const unsigned int NULL_NODE,
                    const unsigned int rootIndex,
                    const float3 *origins,
                    const float3 *directions,
                    const float2 *intervals,
                    const float4 *minBoundHitIndex,
                    const float4 *maxBoundMissIndex,
                    const float4 *firstVertexDominantAxis,
                    bool *stencil,
                    float4 *timeBarycentricsAndTriangleIndex)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  if(stencil[i])
  {
    float3 origin = origins[i];
    float3 dir = directions[i];
    float2 interval = intervals[i];

    float3 invDir = make_float3(1.0f / dir.x,
                                1.0f / dir.y,
                                1.0f / dir.z);
    unsigned int currentNode = rootIndex;
    bool hit = false;
    bool result = false;
    float t = interval.y;;
    unsigned int tri = 0;
    float b1 = -1, b2 = -1;
    float4 minBoundsHit, maxBoundsMiss;
    float4 v0Axis;

    // XXX PERF: it might be possible to eliminate these temporaries
    float tempT, tempB1, tempB2;
    while(currentNode != NULL_NODE)
    {
      minBoundsHit = minBoundHitIndex[currentNode];
      maxBoundsMiss = maxBoundMissIndex[currentNode];

      // leaves (primitives) are listed before interior nodes
      // so bounding boxes occur after the root index
      if(currentNode >= rootIndex)
      {
        hit = intersectBox(origin,
                           invDir,
                           make_float3(minBoundsHit.x,
                                       minBoundsHit.y,
                                       minBoundsHit.z),
                           make_float3(maxBoundsMiss.x,
                                       maxBoundsMiss.y,
                                       maxBoundsMiss.z),
                           interval.x,
                           interval.y);
      } // end if
      else
      {
        v0Axis = firstVertexDominantAxis[currentNode];

        hit = cudaWaldBikkerIntersection
          (origin,
           dir,
           interval.x, interval.y,
           make_float3(v0Axis.x,
                       v0Axis.y,
                       v0Axis.z),
           make_float3(minBoundsHit.x,
                       minBoundsHit.y,
                       minBoundsHit.z),
           __float_as_int(v0Axis.w),
           maxBoundsMiss.x, maxBoundsMiss.y,
           maxBoundsMiss.z, maxBoundsMiss.w,
           tempT, tempB1, tempB2);
        result |= hit;

        // XXX we could potentially merge t and tMax into a single word
        //     as they serve essentially the same purpose
        if(hit)
        {
          t = tempT;
          interval.y = tempT;
          tri = currentNode;
          b1 = tempB1;
          b2 = tempB2;
        } // end if

        // ensure that the miss and hit indices are the same
        // at this point
        maxBoundsMiss.w = minBoundsHit.w;
        hit = false;
      } // end else

      currentNode = hit ? __float_as_int(minBoundsHit.w) : __float_as_int(maxBoundsMiss.w);
    } // end while

    // write results
    stencil[i] = result;
    timeBarycentricsAndTriangleIndex[i] = make_float4(t, b1, b2, __int_as_float(tri));
  } // end if
} // end rbk()

void cudaRayTriangleBVHIntersection(const unsigned int NULL_NODE,
                                    const unsigned int rootIndex,
                                    const float3 *origins,
                                    const float3 *directions,
                                    const float2 *intervals,
                                    const float4 *minBoundHitIndex,
                                    const float4 *maxBoundMissIndex,
                                    const float4 *firstVertexDominantAxis,
                                    bool *stencil,
                                    float4 *timeBarycentricsAndTriangleIndex,
                                    const size_t n)
{
  unsigned int BLOCK_SIZE = 192;
  unsigned int gridSize = n/BLOCK_SIZE;

  if(gridSize)
    rbk<<<gridSize,BLOCK_SIZE>>>(NULL_NODE,
                                 rootIndex,
                                 origins,
                                 directions,
                                 intervals,
                                 minBoundHitIndex,
                                 maxBoundMissIndex,
                                 firstVertexDominantAxis,
                                 stencil,
                                 timeBarycentricsAndTriangleIndex);
  if(n%BLOCK_SIZE)
    rbk<<<1,n%BLOCK_SIZE>>>(NULL_NODE,
                            rootIndex,
                            origins + gridSize*BLOCK_SIZE,
                            directions + gridSize*BLOCK_SIZE,
                            intervals + gridSize*BLOCK_SIZE,
                            minBoundHitIndex,
                            maxBoundMissIndex,
                            firstVertexDominantAxis,
                            stencil + gridSize*BLOCK_SIZE,
                            timeBarycentricsAndTriangleIndex + gridSize*BLOCK_SIZE);
} // end cudaRayTriangleBVHIntersection()

__global__ void shadowKernel(const unsigned int NULL_NODE,
                             const unsigned int rootIndex,
                             const float3 *rayOrigins,
                             const float3 *rayDirections,
                             const float2 *rayIntervals,
                             const float4 *minBoundHitIndex,
                             const float4 *maxBoundMissIndex,
                             const float4 *firstVertexDominantAxis,
                             const bool *stencil,
                             bool *results)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // threads which do no work are assumed to have their
  // rays blocked
  int result = 0;

  if(stencil[i])
  {
    // initially assume no triangle blocks the ray
    result = 1;

    float3 origin = rayOrigins[i];
    float3 dir = rayDirections[i];
    float2 interval = rayIntervals[i];

    float3 invDir = make_float3(1.0f / dir.x,
                                1.0f / dir.y,
                                1.0f / dir.z);
    unsigned int currentNode = rootIndex;
    bool hit = false;
    float4 minBoundsHit, maxBoundsMiss;
    float4 v0Axis;

    // XXX PERF: it might be possible to eliminate these temporaries
    float tempT, tempB1, tempB2;
    while(currentNode != NULL_NODE)
    {
      minBoundsHit = minBoundHitIndex[currentNode];
      maxBoundsMiss = maxBoundMissIndex[currentNode];

      // leaves (primitives) are listed before interior nodes
      // so bounding boxes occur after the root index
      if(currentNode >= rootIndex)
      {
        hit = intersectBox(origin,
                           invDir,
                           make_float3(minBoundsHit.x,
                                       minBoundsHit.y,
                                       minBoundsHit.z),
                           make_float3(maxBoundsMiss.x,
                                       maxBoundsMiss.y,
                                       maxBoundsMiss.z),
                           interval.x,
                           interval.y);
      } // end if
      else
      {
        v0Axis = firstVertexDominantAxis[currentNode];

        hit = cudaWaldBikkerIntersection
          (origin,
           dir,
           interval.x, interval.y,
           make_float3(v0Axis.x,
                       v0Axis.y,
                       v0Axis.z),
           make_float3(minBoundsHit.x,
                       minBoundsHit.y,
                       minBoundsHit.z),
           __float_as_int(v0Axis.w),
           maxBoundsMiss.x, maxBoundsMiss.y,
           maxBoundsMiss.z, maxBoundsMiss.w,
           tempT, tempB1, tempB2);

        if(hit)
        {
          // blocked
          result = 0;
          minBoundsHit.w = __int_as_float(NULL_NODE);
        } // end if

        // ensure that the miss and hit indices are the same
        // at this point
        maxBoundsMiss.w = minBoundsHit.w;
      } // end else

      currentNode = hit ? __float_as_int(minBoundsHit.w) : __float_as_int(maxBoundsMiss.w);
    } // end while
  } // end if

  // write results
  results[i] = result;
} // end shadowKernel()

void cudaShadowRayTriangleBVHIntersectionWithStencil(const unsigned int NULL_NODE,
                                                     const unsigned int rootIndex,
                                                     const float3* rayOrigins,
                                                     const float3* rayDirections,
                                                     const float2* rayIntervals,
                                                     const float4* minBoundHitIndex,
                                                     const float4* maxBoundMissIndex,
                                                     const float4* firstVertexDominantAxis,
                                                     const bool *stencil,
                                                     bool *results,
                                                     const size_t n)
{
  unsigned int BLOCK_SIZE = 192;
  unsigned int gridSize = n/BLOCK_SIZE;

  if(gridSize)
    shadowKernel<<<gridSize,BLOCK_SIZE>>>(NULL_NODE,
                                          rootIndex,
                                          rayOrigins,
                                          rayDirections,
                                          rayIntervals,
                                          minBoundHitIndex,
                                          maxBoundMissIndex,
                                          firstVertexDominantAxis,
                                          stencil,
                                          results);
  if(n%BLOCK_SIZE)
    shadowKernel<<<1,n%BLOCK_SIZE>>>(NULL_NODE,
                                     rootIndex,
                                     rayOrigins + gridSize*BLOCK_SIZE,
                                     rayDirections + gridSize*BLOCK_SIZE,
                                     rayIntervals + gridSize*BLOCK_SIZE,
                                     minBoundHitIndex,
                                     maxBoundMissIndex,
                                     firstVertexDominantAxis,
                                     stencil + gridSize*BLOCK_SIZE,
                                     results + gridSize*BLOCK_SIZE);
} // end cudaRayTriangleBVHIntersection()

__global__ void shadowKernel(const unsigned int NULL_NODE,
                             const unsigned int rootIndex,
                             const float3 *rayOrigins,
                             const float3 *rayDirections,
                             const float2 rayInterval,
                             const float4 *minBoundHitIndex,
                             const float4 *maxBoundMissIndex,
                             const float4 *firstVertexDominantAxis,
                             const bool *stencil,
                             bool *results)
{
  int i = blockIdx.x * blockDim.x + threadIdx.x;

  // threads which do no work are assumed to have their
  // rays blocked
  int result = 0;

  if(stencil[i])
  {
    // initially assume no triangle blocks the ray
    result = 1;

    float3 origin = rayOrigins[i];
    float3 dir = rayDirections[i];

    float3 invDir = make_float3(1.0f / dir.x,
                                1.0f / dir.y,
                                1.0f / dir.z);
    unsigned int currentNode = rootIndex;
    bool hit = false;
    float4 minBoundsHit, maxBoundsMiss;
    float4 v0Axis;

    // XXX PERF: it might be possible to eliminate these temporaries
    float tempT, tempB1, tempB2;
    while(currentNode != NULL_NODE)
    {
      minBoundsHit = minBoundHitIndex[currentNode];
      maxBoundsMiss = maxBoundMissIndex[currentNode];

      // leaves (primitives) are listed before interior nodes
      // so bounding boxes occur after the root index
      if(currentNode >= rootIndex)
      {
        hit = intersectBox(origin,
                           invDir,
                           make_float3(minBoundsHit.x,
                                       minBoundsHit.y,
                                       minBoundsHit.z),
                           make_float3(maxBoundsMiss.x,
                                       maxBoundsMiss.y,
                                       maxBoundsMiss.z),
                           rayInterval.x,
                           rayInterval.y);
      } // end if
      else
      {
        v0Axis = firstVertexDominantAxis[currentNode];

        hit = cudaWaldBikkerIntersection
          (origin,
           dir,
           rayInterval.x, rayInterval.y,
           make_float3(v0Axis.x,
                       v0Axis.y,
                       v0Axis.z),
           make_float3(minBoundsHit.x,
                       minBoundsHit.y,
                       minBoundsHit.z),
           __float_as_int(v0Axis.w),
           maxBoundsMiss.x, maxBoundsMiss.y,
           maxBoundsMiss.z, maxBoundsMiss.w,
           tempT, tempB1, tempB2);

        if(hit)
        {
          // blocked
          result = 0;
          minBoundsHit.w = __int_as_float(NULL_NODE);
        } // end if

        // ensure that the miss and hit indices are the same
        // at this point
        maxBoundsMiss.w = minBoundsHit.w;
      } // end else

      currentNode = hit ? __float_as_int(minBoundsHit.w) : __float_as_int(maxBoundsMiss.w);
    } // end while
  } // end if

  // write results
  results[i] = result;
} // end shadowKernel()

void cudaShadowRayTriangleBVHIntersectionWithStencil(const unsigned int NULL_NODE,
                                                     const unsigned int rootIndex,
                                                     const float3* rayOrigins,
                                                     const float3* rayDirections,
                                                     const float2 &rayInterval,
                                                     const float4* minBoundHitIndex,
                                                     const float4* maxBoundMissIndex,
                                                     const float4* firstVertexDominantAxis,
                                                     const bool *stencil,
                                                     bool *results,
                                                     const size_t n)
{
  unsigned int BLOCK_SIZE = 192;
  unsigned int gridSize = n/BLOCK_SIZE;

  if(gridSize)
    shadowKernel<<<gridSize,BLOCK_SIZE>>>(NULL_NODE,
                                          rootIndex,
                                          rayOrigins,
                                          rayDirections,
                                          rayInterval,
                                          minBoundHitIndex,
                                          maxBoundMissIndex,
                                          firstVertexDominantAxis,
                                          stencil,
                                          results);
  if(n%BLOCK_SIZE)
    shadowKernel<<<1,n%BLOCK_SIZE>>>(NULL_NODE,
                                     rootIndex,
                                     rayOrigins + gridSize*BLOCK_SIZE,
                                     rayDirections + gridSize*BLOCK_SIZE,
                                     rayInterval,
                                     minBoundHitIndex,
                                     maxBoundMissIndex,
                                     firstVertexDominantAxis,
                                     stencil + gridSize*BLOCK_SIZE,
                                     results + gridSize*BLOCK_SIZE);
} // end cudaRayTriangleBVHIntersection()

